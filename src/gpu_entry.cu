#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>
#include <queue>
#include <sys/time.h>
#include "utility.h"

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#else
#include <hip/hip_runtime.h>
#include <cuda_wrappers/algorithm>
#include <cuda_wrappers/new>
#endif

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>

extern "C" {
    #include "photoNs.h"
    #include "gpu_kernel.h" 
}
    #include "gpu_thrust.h" 

#include "gpu_kernel.impl.h"
#include "gpu_func.impl.h"

extern "C" void *task_compute_p2p_gpu(int ntask,
			int* ltask_t, int* ltask_s, int last_time) {
	double t0, t1;
	static int tot_tasks = 0, call_cnt = 0;
	bool first_time = (call_cnt == 0);
	tot_tasks += ntask;
	call_cnt ++;

	P2PPack* pack = packlist;
	pack->ntask = ntask;
	pack->task_t = ltask_t;
	pack->task_s = ltask_s;
	pack->ileaf = first_leaf;
	pack->nleaf = last_leaf - first_leaf;
	pack->ipart = 0;
	pack->npart = NPART;
	pack->rankid = 0;

#ifdef MOREINFO
	long long flopcnt = 0;
	int inode, jnode;
	for(int i = 0; i < pack->ntask; i++ ) 
	{
	    inode = pack->task_t[i];
	    jnode = pack->task_s[i];
	
	    flopcnt += leaf[inode].npart * leaf[jnode].npart;
//	    flopcnt += min(leaf[inode].npart, 128) * min(leaf[jnode].npart, 128);
	}
	LOG(0, "[%d] flop count: %lf (G)\n", PROC_RANK, flopcnt / 1.0e9);
#endif

	double ttest0, ttest1, ttest2, ttest3, ttest4;
	ttest0 = dtime();
	if (first_time) NPACK++;
	devBuffPrep(pack, NPACK);
	ttest1 = dtime();
	
	int nleaf = last_leaf - first_leaf;
	if (pack->nleaf != nleaf || pack->npart != NPART) {
		printf("Pack info error!\n");
		exit(0);
	}

	int rankid = pack->rankid;
	checkDevErrors(devSetDevice(device_ids[rankid]));

	gpuT rrs = 1.0 / splitRadius;
	gpuT coeff = 2.0 * PIisq;

	if (first_time) {
		hostmalloc_in(NPART, nleaf);
		t1 = dtime();
		for (int i=0; i<NPART; ++i) {
//			h_part_pos[i] = part[i].pos[0];
//			h_part_pos[i+NPART] = part[i].pos[1];
//			h_part_pos[i+2*NPART] = part[i].pos[2];
#ifndef INTXYZ
		h_part_pos[i] = part[i].pos[0];
		h_part_pos[i+NPART] = part[i].pos[1];
		h_part_pos[i+2*NPART] = part[i].pos[2];
#else
		h_part_pos[i] = (float) (INT2POS * (part[i].posi[0] - domain_center_i[0]) );
		h_part_pos[i+NPART] = (float)(INT2POS * (part[i].posi[1] - domain_center_i[1]) );
		h_part_pos[i+2*NPART] = (float)(INT2POS * (part[i].posi[2] - domain_center_i[2]) );
#endif







		}
		LOG(3, "[%d] part (NPART %d) reorder time: %lf s\n", PROC_RANK, NPART, dtime()-t1);
		t1 = dtime();
		for (int i=0; i<nleaf; ++i) {
			h_leaf_ip[i] = leaf[i+first_leaf].ipart;
			h_leaf_np[i] = leaf[i+first_leaf].npart;
		}
		LOG(3, "[%d] leaf reorder time: %lf s\n", PROC_RANK, dtime()-t1);
		t1 = dtime();
		devMemcpyHD_part(pack);
		LOG(3, "[%d] copy input time: %lf s\n", PROC_RANK, dtime()-t1);
		hostfree_in();

        	checkDevErrors( devEventCreate(&start_events[rankid]) );
        	checkDevErrors( devEventCreate(&stop_events[rankid]) );
        	checkDevErrors( devEventRecord(start_events[rankid], 0));
        	checkDevErrors( devEventRecord(stop_events[rankid], 0));
	}
	
	if (pack->ntask > 0) {
		t0 = dtime();

		devMemcpyHD_task(pack);
		LOG(3, "[%d] copy task list time: %lf s\n", PROC_RANK, dtime()-t0);

		ttest2 = dtime();

		thrust::sort_by_key(d_keys[rankid].begin(), d_keys[rankid].begin() + pack->ntask, d_values[rankid].begin());
		// reorder <inode, jnode> pairs
		thrust::pair<thrust::device_vector<int>::iterator, thrust::device_vector<int>::iterator> new_end 
		= thrust::reduce_by_key(d_keys[rankid].begin(), d_keys[rankid].begin() + pack->ntask, d_one[rankid].begin(), d_inodes[rankid].begin(), d_nn[rankid].begin());
		thrust::exclusive_scan(d_nn[rankid].begin(), new_end.second, d_in[rankid].begin());

		int numInodes = new_end.second - (d_nn[rankid].begin());
		LOG(3, "[%d]:g%d numInodes: %d NLEAF: %d\n", PROC_RANK, device_ids[rankid], numInodes, nleaf);

		// p2p kernel
#ifdef JSPLIT
		int blockSize = 256;
#else
		int blockSize = MAXLEAF;
#endif
		if (blockSize % WARP != 0) {
			LOG(0, "Error! blockSize for GPU kernel should be mutiple of WARP.\n");
			exit(0);
		}
		int numBlocks = numInodes; //80 * 4;
		// 3 gpuTs for pos (acc), (inode.pos, inode.acc, jnode.pos)
		int sharedMemSize = blockSize * sizeof(gpuT) * 3;
#ifdef __HIPCC__
#ifdef JSPLIT
		if (blockSize % MAXLEAF != 0 || blockSize > 1024) {
			LOG(0, "Error! blockSize (<=1024) for GPU p2p_kernel2 should be mutiple of MAXLEAF.\n");
			exit(0);
		}
		checkKernelErrors((P2P_kernel2<<< numBlocks, blockSize, sharedMemSize>>>(SoftenScale, MASSPART, rrs, coeff, MAXLEAF,
#else
		checkKernelErrors((P2P_kernel3<<< numBlocks, blockSize, sharedMemSize>>>(SoftenScale, MASSPART, rrs, coeff,
#endif 
		numInodes, 
		thrust::raw_pointer_cast(d_inodes[rankid].data()),
		thrust::raw_pointer_cast(d_in[rankid].data()),
		thrust::raw_pointer_cast(d_nn[rankid].data()),
		thrust::raw_pointer_cast(d_values[rankid].data()),
		pack->ipart,
		pack->npart,
		d_part_pos[rankid],
		d_part_buf[rankid], // acc
		d_leaf_ip[rankid],
		d_leaf_np[rankid])));
#else
		hipLaunchKernelGGL(P2P_kernel, numBlocks, blockSize, sharedMemSize, 0, SoftenScale, MASSPART, rrs, coeff,
		numInodes, 
		thrust::raw_pointer_cast(d_inodes[rankid].data()),
		thrust::raw_pointer_cast(d_in[rankid].data()),
		thrust::raw_pointer_cast(d_nn[rankid].data()),
		thrust::raw_pointer_cast(d_values[rankid].data()),
		pack->ipart,
		pack->npart,
		d_part_pos[rankid],
		d_part_buf[rankid], // acc
		d_leaf_ip[rankid],
		d_leaf_np[rankid],
		(uint_8b*)0,
		(uint_8b)0);
#endif
	}

	ttest3 = dtime();

	if (last_time) {
        	checkDevErrors(devEventRecord(stop_events[rankid], 0));
#ifdef TASKLOG
		task_record(0, tot_tasks);	
#endif
                LOG(2, " [%d] ntask = %d (call cnt: %d)\n", PROC_RANK, tot_tasks, call_cnt);
		tot_tasks = 0;
		call_cnt = 0;
	}
	ttest4 = dtime();
#ifdef MOREINFO
	printf("%lf %lf %lf %lf\n", ttest1-ttest0, ttest2-ttest1, ttest3-ttest2, ttest4-ttest3);
#endif
	return NULL;
}

extern "C" void *p2ppack_compute_gpu(P2PPack* packs, int npack) {
	int tot_tasks = 0;
	for (int i=0; i<npack; ++i)
		tot_tasks += packs[i].ntask;
#ifdef TASKLOG
	task_record(0, tot_tasks);	
#endif
        LOG(2, " [%d] ntask = %d (NPACK: %d)\n", PROC_RANK, tot_tasks, npack);
	if (npack == 0)
		return NULL;

	double t0, t1;

	double ttest0, ttest1, ttest2, ttest3, ttest4, ttest5;
	ttest0 = dtime();
	devBuffPrep(packs, npack);
	ttest1 = dtime();

	gpuT rrs = 1.0 / splitRadius;
	gpuT coeff = 2.0 * PIisq;

	int nleaf = last_leaf - first_leaf;
	hostmalloc_in(NPART, nleaf);
	t0 = dtime();
	for (int i=0; i<NPART; ++i) {
//		h_part_pos[i] = part[i].pos[0];
//		h_part_pos[i+NPART] = part[i].pos[1];
//		h_part_pos[i+2*NPART] = part[i].pos[2];
#ifndef INTXYZ
		h_part_pos[i] = part[i].pos[0];
		h_part_pos[i+NPART] = part[i].pos[1];
		h_part_pos[i+2*NPART] = part[i].pos[2];

#ifdef TEST_FLOAT
		float dispf = 2000000.0;
		
		h_part_pos[i] += dispf;
		h_part_pos[i+NPART] += dispf;
		h_part_pos[i+2*NPART] += dispf;
#endif

#else
	//	h_part_pos[i] = (float) (INT2POS * part[i].posi[0]);
	//	h_part_pos[i+NPART] = (float)(INT2POS * part[i].posi[1]);
	//	h_part_pos[i+2*NPART] = (float)(INT2POS *  part[i].posi[2]);

		h_part_pos[i] = (float) (INT2POS * (part[i].posi[0] - domain_center_i[0]) );
		h_part_pos[i+NPART] = (float)(INT2POS * (part[i].posi[1] - domain_center_i[1]) );
		h_part_pos[i+2*NPART] = (float)(INT2POS * (part[i].posi[2] - domain_center_i[2]) );

#endif




	}
	LOG(3, "[%d] part (NPART %d) reorder time: %lf s\n", PROC_RANK, NPART, dtime()-t0);
	t0 = dtime();
	for (int i=0; i<nleaf; ++i) {
		h_leaf_ip[i] = leaf[i+first_leaf].ipart;
		h_leaf_np[i] = leaf[i+first_leaf].npart;
	}
	LOG(3, "[%d] leaf reorder time: %lf s\n", PROC_RANK, dtime()-t0);

	int* numInodes_arr = (int*)malloc(sizeof(int) * npack);

#pragma omp parallel for num_threads(npack)
	for (int n=0; n<npack; ++n) {
		if (packs[n].ntask == 0)
			continue;
		int i = packs[n].rankid;
 
		checkDevErrors(devSetDevice(device_ids[i]));

		t1 = dtime();
		devMemcpyHD_task(packs, n);
		LOG(3, "[%d] copy task list %d time: %lf s\n", PROC_RANK, n, dtime()-t1);
	
		t1 = dtime();
		devMemcpyHD_part(packs, n);
		LOG(3, "[%d] copy input time: %lf s\n", PROC_RANK, dtime()-t1);

		thrust::sort_by_key(d_keys[i].begin() + off_ntask[n], d_keys[i].begin() + off_ntask[n] + packs[n].ntask, d_values[i].begin() + off_ntask[n]);
		// reorder <inode, jnode> pairs
		thrust::pair<thrust::device_vector<int>::iterator, thrust::device_vector<int>::iterator> new_end 
		= thrust::reduce_by_key(d_keys[i].begin() + off_ntask[n], d_keys[i].begin() + off_ntask[n] + packs[n].ntask, d_one[i].begin() + off_ntask[n], d_inodes[i].begin() + off_nleaf[n], d_nn[i].begin() + off_nleaf[n]);
		thrust::exclusive_scan(d_nn[i].begin() + off_nleaf[n], new_end.second, d_in[i].begin() + off_nleaf[n]);
		numInodes_arr[n] = new_end.second - (d_nn[i].begin() + off_nleaf[n]);
	}
	hostfree_in();
	ttest2 = dtime();

	for (int i=0; i<device_cnt; ++i) {
		checkDevErrors(devSetDevice(device_ids[i]));
        	checkDevErrors( devEventCreate(&start_events[i]) );
        	checkDevErrors( devEventCreate(&stop_events[i]) );
        	checkDevErrors( devEventRecord(start_events[i], 0));
        	checkDevErrors( devEventRecord(stop_events[i], 0));
	}

#pragma omp parallel for num_threads(npack)
	for (int n=0; n<npack; ++n) {
		if (packs[n].ntask == 0)
			continue;
		int i = packs[n].rankid;

		checkDevErrors(devSetDevice(device_ids[i]));

		int numInodes = numInodes_arr[n]; 

		LOG(3, "[%d]:g%d numInodes: %d NLEAF: %d\n", PROC_RANK, device_ids[i], numInodes, packs[n].nleaf);
		// p2p kernel
#ifdef JSPLIT
		int blockSize = 256;
#else
		int blockSize = MAXLEAF;
#endif
		if (blockSize % WARP != 0) {
			LOG(0, "Error! blockSize for GPU kernel should be mutiple of WARP.\n");
			exit(0);
		}
		int numBlocks = numInodes; //80 * 4;
		// 3 gpuTs for pos (acc), (inode.pos, inode.acc, jnode.pos)
		int sharedMemSize = blockSize * sizeof(gpuT) * 3;
#ifdef __HIPCC__
#ifdef JSPLIT
		if (blockSize % MAXLEAF != 0 || blockSize > 1024) {
			LOG(0, "Error! blockSize (<=1024) for GPU p2p_kernel2 should be mutiple of MAXLEAF.\n");
			exit(0);
		}
		checkKernelErrors((P2P_kernel2<<< numBlocks, blockSize, sharedMemSize>>>(SoftenScale, MASSPART, rrs, coeff, MAXLEAF,
#else
		checkKernelErrors((P2P_kernel3<<< numBlocks, blockSize, sharedMemSize>>>(SoftenScale, MASSPART, rrs, coeff,
#endif 
			numInodes, 
			thrust::raw_pointer_cast(d_inodes[i].data() + off_nleaf[n]),
			thrust::raw_pointer_cast(d_in[i].data() + off_nleaf[n]),
			thrust::raw_pointer_cast(d_nn[i].data() + off_nleaf[n]),
			thrust::raw_pointer_cast(d_values[i].data() + off_ntask[n]),
			packs[n].ipart,
			packs[n].npart,
			d_part_pos[i] + off_npart[n]*3,
			d_part_buf[i] + off_npart[n]*3, // acc
			d_leaf_ip[i] + off_nleaf[n],
			d_leaf_np[i] + off_nleaf[n])));
#else
		hipLaunchKernelGGL(P2P_kernel, numBlocks, blockSize, sharedMemSize, 0, SoftenScale, MASSPART, rrs, coeff,
			numInodes, 
			thrust::raw_pointer_cast(d_inodes[i].data() + off_nleaf[n]),
			thrust::raw_pointer_cast(d_in[i].data() + off_nleaf[n]),
			thrust::raw_pointer_cast(d_nn[i].data() + off_nleaf[n]),
			thrust::raw_pointer_cast(d_values[i].data() + off_ntask[n]),
			packs[n].ipart,
			packs[n].npart,
			d_part_pos[i] + off_npart[n]*3,
			d_part_buf[i] + off_npart[n]*3, // acc
			d_leaf_ip[i] + off_nleaf[n],
			d_leaf_np[i] + off_nleaf[n],
			(uint_8b*)0,
			(uint_8b)0);
#endif
	}
	ttest3 = dtime();

	for (int i=0; i<device_cnt; ++i) {
		checkDevErrors(devSetDevice(device_ids[i]));
        	checkDevErrors(devEventRecord(stop_events[i], 0));
	}

	free(numInodes_arr);
#ifdef MOREINFO
	printf("%lf %lf %lf\n", ttest1-ttest0, ttest2-ttest1, ttest3-ttest2);
#endif

	return NULL;
}

#include "gpu_active.impl.h"
